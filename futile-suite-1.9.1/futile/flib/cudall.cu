#include <iostream>
#include <stdio.h>
#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hip/hip_runtime_api.h"
#include "config.h"
 
#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8


static const char *_cublasGetErrorString(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
        case HIPBLAS_STATUS_NOT_SUPPORTED:
            return "HIPBLAS_STATUS_NOT_SUPPORTED";
#if CUDA_VERSION >= 6500
        case HIPBLAS_STATUS_UNKNOWN:
            return "HIPBLAS_STATUS_UNKNOWN";
#endif
    }
    return "<unknown>";
}

static const char *_cufftGetErrorString(hipfftResult error)
{
    switch (error)
    {
        case HIPFFT_SUCCESS:
            return "HIPFFT_SUCCESS";
        case HIPFFT_INVALID_PLAN:
            return "HIPFFT_INVALID_PLAN";
        case HIPFFT_ALLOC_FAILED:
            return "HIPFFT_ALLOC_FAILED";
        case HIPFFT_INVALID_TYPE:
            return "HIPFFT_INVALID_TYPE";
        case HIPFFT_INVALID_VALUE:
            return "HIPFFT_INVALID_VALUE";
        case HIPFFT_INTERNAL_ERROR:
            return "HIPFFT_INTERNAL_ERROR";
        case HIPFFT_EXEC_FAILED:
            return "HIPFFT_EXEC_FAILED";
        case HIPFFT_SETUP_FAILED:
            return "HIPFFT_SETUP_FAILED";
        case HIPFFT_INVALID_SIZE:
            return "HIPFFT_INVALID_SIZE";
        case HIPFFT_UNALIGNED_DATA:
            return "HIPFFT_UNALIGNED_DATA";
    }
    return "<unknown>";
}


hipStream_t stream1=NULL;
hipblasHandle_t handle1=NULL;
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


#define cufftErrchk(ans) { __cufftAssert((ans), __FILE__, __LINE__); }

inline void __cufftAssert(hipfftResult code, const char *file, const int line, bool abort=true)
{
   if(code != HIPFFT_SUCCESS) 
   {
      fprintf(stderr, "cufftAssert : %s %s %d.\n",
      _cufftGetErrorString(code), file, line);
      if (abort) exit(-1);
   }
}

#define cublasErrchk(ans) { __cublasAssert((ans), __FILE__, __LINE__); }

inline void __cublasAssert(hipblasStatus_t code, const char *file, const int line, bool abort=true)
{
   if(code !=HIPBLAS_STATUS_SUCCESS) 
   {
      fprintf(stderr, "cublasAssert : %s %s %d.\n",
      _cublasGetErrorString(code), file, line);
      if (abort) exit(-1);
   }
}

// synchronize blocks
extern "C" void FC_FUNC(synchronize, SYNCHRONIZE)() {
   hipStreamSynchronize(stream1);
}


// allocate device memory
extern "C" void FC_FUNC(cudamalloc, CUDAMALLOC) (int *size, Real **d_data,int *ierr) {

  *ierr = hipMalloc((void**)d_data, sizeof(Real)*(*size));
  //errors should be treated in the fortran part
}

// allocate device memory
extern "C" void FC_FUNC(cudamemset, CUDAMEMSET) (Real **d_data, int* value, int* size,int *ierr) {

  *ierr = hipMemsetAsync((void*)*d_data, *value, sizeof(Real)*(*size),stream1);
}

extern "C" void FC_FUNC(cudafree, CUDAFREE) (Real **d_data) {
  hipFree(*d_data);
}

// set device memory
extern "C" void FC_FUNC_(reset_gpu_data, RESET_GPU_DATA)(int *size, Real* h_data, Real **d_data){
  hipMemcpyAsync(*d_data, h_data, sizeof(Real)*(*size),
         hipMemcpyHostToDevice,stream1);
  gpuErrchk( hipPeekAtLastError() );
}

// copy data on the card
extern "C" void FC_FUNC_(copy_gpu_data, COPY_GPU_DATA)(int *size, Real** dest_data, Real **send_data){
  hipMemcpyAsync(*dest_data, *send_data, sizeof(Real)*(*size),
         hipMemcpyDeviceToDevice,stream1);
  gpuErrchk( hipPeekAtLastError() );
}


// read device memory
extern "C" void FC_FUNC_(get_gpu_data, GET_GPU_DATA)(int *size, Real *h_data, Real **d_data) {
  hipMemcpyAsync(h_data, *d_data, sizeof(Real)*(*size),
         hipMemcpyDeviceToHost,stream1);
  gpuErrchk( hipPeekAtLastError() );
}

extern "C" void FC_FUNC_(cuda_get_mem_info, CUDA_GET_MEM_INFO)(size_t* freeSize, size_t* totalSize){
 gpuErrchk(hipMemGetInfo(freeSize,totalSize));
}

// set device memory
extern "C" void FC_FUNC_(poisson_cublas_daxpy, POISSON_CUBLAS_DAXPY)(int *size, const double* alpha, Real** d_x,int* facx, Real ** d_y, int* facy,int* offset_y){

  hipblasSetStream(handle1, stream1);
  cublasErrchk(hipblasDaxpy(handle1,*size,alpha,*d_x,*facx,*d_y+*offset_y,*facy));
//  gpuErrchk( hipPeekAtLastError() );
}


extern "C" void FC_FUNC_(cudagetdevicecount, CUDAGETDEVICECOUNT)(int* num_devices){
  gpuErrchk(hipGetDeviceCount(num_devices));
}

extern "C" void FC_FUNC_(cudasetdevice, CUDASETDEVICE)(int* device){
  gpuErrchk(hipSetDevice(*device));
}

extern "C" void FC_FUNC_(cudadevicereset, CUDADEVICERESET)(int* device){
  gpuErrchk(hipDeviceReset());
}

